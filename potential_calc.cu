#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <sys/time.h>
// #include <cutil_inline.h>
#include "Vector3.hpp"
#include "mydefs.hpp"

// #define SDATA(index)      cutilBankChecker(sdata, index)
// #define SDATA(index)      sdata[index]

// Print a message if a CUDA error occurred
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if(hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Kernel definition (2nd version)
__global__ void
calc_potential_exact_kernel(   const fptype *charge_gmem,
                                const int xdim, const int ydim, const int zdim,
                                fptype *potential_gmem  )
{
    const int N = xdim*ydim*zdim;
    int bi = blockIdx.x;
    int stride = ceilf(N / (fptype)blockDim.x);
    int ti1 = threadIdx.x * stride;

    if(bi >= N) // if block exceeds, don't proceed (unnecessary check)
        return;

// reset shared memory
    extern __shared__ fptype sdata[];
    sdata[threadIdx.x] = 0;
    // __syncthreads;

    if(ti1 < N) // calculate potential only if start of thread doesn't exceed
    {
        __shared__ int xs, ys, zs; // target coords per block
        if(threadIdx.x == 0) {
            xs = bi % xdim;
            ys = ((bi - xs) / xdim) % ydim;
            zs = (bi - xs - ys*xdim) / (xdim*ydim);
        }
        __syncthreads();
        int x = xs;
        int y = ys;
        int z = zs;

        fptype pot = 0;
        // int i = ti1;
        for(int i = ti1; i < ti1 + stride; i++)
        {
            if(i < N) { // calculate potential only if this point doesn't exceed
                fptype q = charge_gmem[i];
                // __syncthreads();
                // if(q != 0)
                {
                    int x_ = i % xdim; // source coords per thread
                    int y_ = ((i - x_) / xdim) % ydim;
                    int z_ = (i - x_ - y_*xdim) / (xdim*ydim);
                    // potential due this thread's charge
                    if(bi != i) { // skip on itself to avoid div by zero
                        // Vector3 V(x-x_, y-y_, z-z_);
                        // fptype dist = V.magnitude();
                        fptype R = sqrtf((x-x_)*(x-x_) + (y-y_)*(y-y_) + (z-z_)*(z-z_));
                        pot += q / R;
                    }
                }
            }
        }
        sdata[threadIdx.x] = pot;
    }
    __syncthreads();

// parallel reduction to sum up potential from threads
// (must use all 1024 threads)
    if (threadIdx.x < 512)
        sdata[threadIdx.x] += sdata[threadIdx.x + 512];
    __syncthreads();
    if (threadIdx.x < 256)
        sdata[threadIdx.x] += sdata[threadIdx.x + 256];
    __syncthreads();
    if (threadIdx.x < 128)
        sdata[threadIdx.x] += sdata[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64)
        sdata[threadIdx.x] += sdata[threadIdx.x + 64];
    __syncthreads();
    if (threadIdx.x < 32)
        sdata[threadIdx.x] += sdata[threadIdx.x + 32];
    __syncthreads();
    if (threadIdx.x < 16)
        sdata[threadIdx.x] += sdata[threadIdx.x + 16];
    __syncthreads();
    if (threadIdx.x < 8)
        sdata[threadIdx.x] += sdata[threadIdx.x + 8];
    __syncthreads();
    if (threadIdx.x < 4)
        sdata[threadIdx.x] += sdata[threadIdx.x + 4];
    __syncthreads();
    if (threadIdx.x < 2)
        sdata[threadIdx.x] += sdata[threadIdx.x + 2];
    __syncthreads();
    if (threadIdx.x < 1)
        sdata[threadIdx.x] += sdata[threadIdx.x + 1];
    __syncthreads();
// write summed potential to global memory
    if(threadIdx.x == 0)
        potential_gmem[bi] = sdata[0];
}



// Exact O(N^2) calculation of potential
int calc_potential_exact_gpu( const fptype *charge,
                        const int xdim, const int ydim, const int zdim,
                        fptype *potential)
{

    int status = 0;
    static int first_time = 1;

    // set up device memory pointers
    static fptype *charge_gmem = NULL;
    static fptype *potential_gmem = NULL;

    if(first_time) {
        // select device to use
        hipSetDevice(1);
        // allocate memory on device
        hipMalloc((void**)&charge_gmem,    zdim*ydim*xdim * sizeof(fptype));
        checkCUDAError("Allocate charge_gmem");
        hipMalloc((void**)&potential_gmem, zdim*ydim*xdim * sizeof(fptype));
        checkCUDAError("Allocate potential_gmem");
        if(charge_gmem == NULL || potential_gmem == NULL) {
            fprintf(stderr, "%s:%d Error allocating memory on GPU\n", __FILE__, __LINE__);
            return EXIT_FAILURE;
        }
    }

    // int currentDevice;
    // hipGetDevice(&currentDevice);
    // printf("using device %d\n", currentDevice);

    // copy charge array to device global memory
    hipMemcpy(charge_gmem, charge, zdim*ydim*xdim * sizeof(fptype), hipMemcpyHostToDevice);
    checkCUDAError("Copying charge_gmem");


    // set up kernel parameters
    int problem_size = zdim*ydim*xdim;
    dim3 grid = problem_size;
    dim3 threads = 1024;

    if(first_time)
        printf("launching kernel with %u blocks and %u threads...\n",
                    grid.x*grid.y*grid.z, threads.x*threads.y*threads.z);

    // start timer
    timeval time1, time2;
    status |= gettimeofday(&time1, NULL);

    // launch the kernel
    calc_potential_exact_kernel <<<grid, threads, 1024 * sizeof(fptype)>>>
        (charge_gmem, xdim, ydim, zdim, potential_gmem);
    checkCUDAError("Exeuting Kernel calc_potential_exact_kernel()");
    hipDeviceSynchronize();

    // read the timer
    status |= gettimeofday(&time2, NULL);
    // double deltatime = (time2.tv_sec + time2.tv_usec/1e6) - (time1.tv_sec + time1.tv_usec/1e6);
    // printf("  Kernel completed in %f seconds.\n", deltatime);

    // copy potential (the result of kernel) to host main memory
    hipMemcpy(potential, potential_gmem, zdim*ydim*xdim * sizeof(fptype), hipMemcpyDeviceToHost);
    checkCUDAError("Copying potential_gmem");
    // hipFree(charge_gmem);
    // hipFree(potential_gmem);
    hipDeviceSynchronize();

    first_time = 0;
    return EXIT_SUCCESS;
}
