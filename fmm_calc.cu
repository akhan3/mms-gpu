#include "hip/hip_runtime.h"
#ifdef _OPENMP
#include <omp.h>
#endif
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <sys/time.h>
#include <cmath>
#include "Box.hpp"
#include "Queue.hpp"
#include "Cmpx.hpp"
#include "Vector3.hpp"
#include "numerics.hpp"
#include "helper_functions.hpp"


// FMM algorithm in BFS
// ===============================
int fmm_bfs(        const fptype *charge,
                    fptype *potential,
                    fptype *potential_gmem,
                    Box *const root,
                    const unsigned int limit,
                    const unsigned int actual_limit,
                    const int P,    // multipole series truncation (l = 0...P)
                    const int xdim, const int ydim, const int zdim,
                    const int zc,   // charge layer
                    FILE *paniclog,
                    const int use_gpu,
                    const int verbose_level
                )
{
    int status = 0;
    assert(limit == actual_limit);  // unable to support arbitrary depth calculations.
    assert(limit <= actual_limit);
    timeval time1, time2;
    status |= gettimeofday(&time1, NULL);
    if(verbose_level >= 6)
        printf("    Executing FMM algorithm...\n");
    unsigned int prev_level = 0;

    const unsigned int N = (unsigned int)pow(4, limit);
    void **queue_mem = (void**)malloc(N * sizeof(void*));
    if(queue_mem == NULL) {
        fprintf(stderr, "%s:%d Error allocating memory\n", __FILE__, __LINE__);
        return EXIT_FAILURE;
    }
    Queue Q_tree(N, queue_mem);
    Q_tree.enqueue((void*)root);

// timers for profiling
    double t_coeff = 0;
    double t_potential = 0;
    double t_potential_nearest = 0;
    double deltatime = 0;
    timeval t1, t2;

// iterate over all the boxes in tree
    // int total_boxes = (4*xdim*ydim - 1) / 3;
    // for(int b = 0; b < total_boxes; b++)
    // {
        // Box *n = root + b;
    while(!Q_tree.isEmpty())
    {
        Box *n = (Box*)Q_tree.dequeue();
        if(n->level < limit)
            for(int i=0; i<=3; i++)
                Q_tree.enqueue(n->child[i]); // populate queue with children nodes

        if(n->level <= 1)   // no FMM steps for Level-0 and Level-1
            continue;

// function to perform on node
        if(prev_level != n->level) {
            if(prev_level >= 2) {
                status |= gettimeofday(&time2, NULL);
                double deltatime = (time2.tv_sec + time2.tv_usec/1e6) - (time1.tv_sec + time1.tv_usec/1e6);
                status |= gettimeofday(&time1, NULL);
                if(verbose_level >= 10)
                    printf("done in %f seconds.\n", deltatime); fflush(NULL);
            // saving this level potential
                // char filename_pot[200];
                // if     (use_gpu == 0) sprintf(filename_pot, "potential_cpu_L%d.dat", n->level - 1);
                // else if(use_gpu == 1) sprintf(filename_pot, "potential_gpu_L%d.dat", n->level - 1);
                // else if(use_gpu == 2) sprintf(filename_pot, "potential_gpuemu_L%d.dat", n->level - 1);
                // else                  sprintf(filename_pot, "potential_gpugpu_L%d.dat", n->level - 1);
                // status |= save_scalar3d(potential, zdim, ydim, xdim, filename_pot, 100);
                // if(status) return EXIT_FAILURE;
            }
            prev_level = n->level;
            if(verbose_level >= 6) {
                int width = pow(2, actual_limit-n->level);
                printf("    Level%d (%dx%d boxes, size=%dx%d)... ",
                    n->level, (int)pow(2, n->level), (int)pow(2, n->level), width, width);
                fflush(NULL);
            }
        }

        // if(n->is_pruned()) {
            // continue;
        // }

        // char idstring[100];
        // n->get_idstring(idstring);
        // printf("L%d%s(%d,%d)=L%d(%.1f,%.1f) \n", n->level, idstring, n->x, n->y, actual_limit, n->cx, n->cy);


        gettimeofday(&t1, NULL);

        fptype q = 0;

    // Calculate multipole coefficients for the source box
        Cmpx mpc[(P+1)*(P+1)];
        // checking for source charges in the source box
        fptype charge_found = 0;
        fptype width = pow(2, actual_limit-n->level);
        int yy1 = ceil(n->cy-width/2);
        int yy2 = floor(n->cy+width/2);
        #ifdef _OPENMP
        // #pragma omp parallel for
        #endif
        for(int yy=yy1; yy<=yy2; yy++) {
        // for(int yy=ceil(n->cy-width/2); yy<=floor(n->cy+width/2); yy++) {
            for(int xx=ceil(n->cx-width/2); xx<=floor(n->cx+width/2); xx++) {
                q = charge[yy*xdim + xx];
                if(q != 0) { // if charge found
                    charge_found = 1;
                    Cmpx r_(xx - n->cx, yy - n->cy);
                    for(int l=0; l<=P; l++) {
                        for(int m=-l; m<=l; m++) {
                            Cmpx sph = spherical_harmonic(l, m, M_PI/2, r_.get_ang()).conjugate();
                            sph *= q * pow(r_.get_mag(), l);
                            mpc[l*l+l+m] += sph;
                            // mpc[l*l+l+m] += q * pow(r_.get_mag(), l) * spherical_harmonic(l, m, M_PI/2, r_.get_ang()).conjugate();
                        } // m loop
                    } // l loop
                } // if(q != 0)
            } // source charge loop
        } // source charge loop
        // NEWLINE;

        gettimeofday(&t2, NULL);
        deltatime = (t2.tv_sec + t2.tv_usec/1e6) - (t1.tv_sec + t1.tv_usec/1e6);
        t_coeff += deltatime;

        if(! charge_found) {
            // n->prune();
            continue;
        }

        // gettimeofday(&t1, NULL);

        if(charge_found)
        {
            gettimeofday(&t1, NULL);

            // calculation of potential at the boxes in 27 boxes of interaction list
            if(use_gpu && width >= 4) {
                status |= fmm_gpu(  n,
                                    mpc,
                                    potential_gmem, limit, P,
                                    xdim, ydim, zdim, zc,
                                    use_gpu, verbose_level);
            }
            else {
                #ifdef _OPENMP
                // #pragma omp parallel for
                #endif
                for(int i=0; i<27; i++) {
                    Box *ni = n->interaction[i];
                    if(ni != NULL) {
                        for(int yy=ceil(ni->cy-width/2); yy<=floor(ni->cy+width/2); yy++) {
                            for(int xx=ceil(ni->cx-width/2); xx<=floor(ni->cx+width/2); xx++) {
                                for (int zp = 0; zp < zdim; zp++) { // for each potential layer in zdim
                                    Vector3 r(xx - n->cx, yy - n->cy, zp - zc);
                                    Cmpx sum_over_lm;
                                    for(int l=0; l<=P; l++) {
                                        Cmpx sum_over_m;
                                        for(int m=-l; m<=l; m++) {
                                            Cmpx sph = spherical_harmonic(l, m, r.colatitude(), r.azimuth());
                                            sph *= (1.0*factorial(l-abs(m))) / factorial(l+abs(m));
                                            sph *= mpc[l*l+l+m];
                                            sum_over_m += sph;
                                            // sum_over_m += (1.0*factorial(l-abs(m))) / factorial(l+abs(m)) * mpc[l*l+l+m] * spherical_harmonic(l, m, r.colatitude(), r.azimuth());
                                        }
                                        sum_over_m *= 1 / pow(r.magnitude(), l+1);
                                        sum_over_lm += sum_over_m;
                                        // sum_over_lm += 1 / pow(r.magnitude(), l+1) * sum_over_m;
                                    }
                                    potential[zp*ydim*xdim + yy*xdim + xx] += sum_over_lm.get_re();
                                    // potential[yy*xdim+xx] += (sum_over_lm.get_re() > 0) ? sum_over_lm.get_mag() : -sum_over_lm.get_mag();

                                    // const fptype threshold = 1e-2;
                                    // fptype modangle = fabs(sum_over_lm.get_ang());
                                    // modangle = (modangle < M_PI-modangle) ? modangle : M_PI-modangle;
                                    // if(modangle > threshold) {
                                        // if(verbose_level >= 0)
                                            // printf("PANIC!! L%d   R=%g   angle=%g\n", n->level, r.magnitude(), modangle);
                                        // fprintf(paniclog, "%d   %g   %g\n", n->level, r.magnitude(), modangle);
                                    // }
                                } // potential layers
                            }
                        }
                    } // if(ni != NULL)
                } // interaction loop
            } // if(! use_gpu)

            gettimeofday(&t2, NULL);
            deltatime = (t2.tv_sec + t2.tv_usec/1e6) - (t1.tv_sec + t1.tv_usec/1e6);
            t_potential += deltatime;

            // calculation with neighbor list at the deepest level
            if(n->level == actual_limit) {
                gettimeofday(&t1, NULL);
                // printf("nearest potential calulcation.\n");
                // assert(n->cx == n->x && n->cy == n->y);
                // fptype q_prev = q;
                // q = charge[(int)(n->cy*xdim + n->cx)];
                // assert(q == q_prev);

                for (int zp = 0; zp < zdim; zp++) { // for each potential layer in zdim
                    if(zp != zc) { // neighbor on other layers at self position
                        Vector3 r(0, 0, zp - zc);
                        potential[zp*ydim*xdim + (int)(n->cy*xdim + n->cx)] += q / r.magnitude();
                    }
                    for(int i=0; i<8; i++) {
                        Box *nb = n->neighbor[i];
                        if(nb != NULL) {
                            Vector3 r(nb->cx - n->cx, nb->cy - n->cy, zp - zc);
                            potential[zp*ydim*xdim + (int)(nb->cy*xdim + nb->cx)] += q / r.magnitude();
                        }
                    } // neighbor loop

                    gettimeofday(&t2, NULL);
                    deltatime = (t2.tv_sec + t2.tv_usec/1e6) - (t1.tv_sec + t1.tv_usec/1e6);
                    t_potential_nearest += deltatime;
                    // printf("nearest potential calulcation took %f seconds so far.\n", t_potential_nearest);

                } // for each potential layer in zdim
            } // if deepest level

        } // if(charge_found)

        // gettimeofday(&t2, NULL);
        // deltatime = (t2.tv_sec + t2.tv_usec/1e6) - (t1.tv_sec + t1.tv_usec/1e6);
        // t_potential += deltatime;

    } // while(!Q_tree.isEmpty())

    status |= gettimeofday(&time2, NULL);
    deltatime = (time2.tv_sec + time2.tv_usec/1e6) - (time1.tv_sec + time1.tv_usec/1e6);
    if(verbose_level >= 10)
        printf("done in %f seconds.\n", deltatime); fflush(NULL);
    // saving this level potential
    // char filename_pot[200];
    // if     (use_gpu == 0) sprintf(filename_pot, "potential_cpu_L%d.dat", limit);
    // else if(use_gpu == 1) sprintf(filename_pot, "potential_gpu_L%d.dat", limit);
    // else if(use_gpu == 2) sprintf(filename_pot, "potential_gpuemu_L%d.dat", limit);
    // else                  sprintf(filename_pot, "potential_gpugpu_L%d.dat", limit);
    // status |= save_scalar3d(potential, zdim, ydim, xdim, filename_pot, 100);
    // if(status) return EXIT_FAILURE;
    // if(verbose_level >= 10) {
        // printf("done in %f seconds.\n", deltatime);
        // printf("FMM coeff calulcation took %f seconds.\n", t_coeff);
        // printf("FMM potential calulcation took %f seconds.\n", t_potential);
        // printf("nearest potential calulcation took %f seconds.\n", t_potential_nearest);
    // }

    free(queue_mem);
    return status ? EXIT_FAILURE : EXIT_SUCCESS;
}





int fmm_calc(   const fptype *charge,
                fptype *potential,
                const int xdim, const int ydim, const int zdim,
                const int P,    // multipole series truncation (l = 0...P)
                const int use_gpu,
                const int verbose_level )
{
    static int first_time = 1;
    int status = 0;
    const unsigned int logN = ceil(log2f(xdim * ydim) / log2f(4));
    // FILE *paniclog = fopen("paniclog.dat", "w");
    FILE *paniclog = fopen("paniclog.dat", "a");
    fprintf(paniclog, "# FMM: New run\n");

    timeval time1, time2;
    double deltatime;
    status |= gettimeofday(&time1, NULL);

// allocate memory for the Tree and its associated BFS Queue
    int total_boxes = (4*xdim*ydim - 1) / 3;
    printf("sizeof(Box) = %lu\n", sizeof(Box));
    printf("total Boxes in the tree = %d\n", total_boxes);
    printf("memory required for the tree = %lu Bytes\n", total_boxes * sizeof(Box));
    // printf("memory required for the tree = %.0f KB\n", total_boxes * sizeof(Box) / 1024.0);
    Box *tree = (Box*)malloc(total_boxes * sizeof(Box));
    int len = xdim * ydim;
    // void **contents_ = new void*[len]();
    void **queue_mem = (void**)malloc(len * sizeof(void*));
    if(tree == NULL || queue_mem == NULL) {
        fprintf(stderr, "%s:%d Error allocating memory\n", __FILE__, __LINE__);
        return EXIT_FAILURE;
    }

// generate the tree
    tree[0] = Box(0, 0, logN);
    Box *root = &tree[0];
    // Box *root = new Box(0, 0, logN);
    // root->create_tree_recurse(logN);
    root->create_tree_bfs(logN, queue_mem);
    root->find_neighbors_recurse(root, logN);
    free(queue_mem);

    status |= gettimeofday(&time2, NULL);
    deltatime = (time2.tv_sec + time2.tv_usec/1e6) - (time1.tv_sec + time1.tv_usec/1e6);
    if(verbose_level >= 0)
        printf("Tree: took %f seconds\n", deltatime);
    fflush(NULL);


    // timeval time1, time2;
    status |= gettimeofday(&time1, NULL);

// pinned host memory and associated device memory
    static fptype *potential_pinned = NULL;
    static fptype *potential_gmem = NULL;
    if(first_time) {
        hipHostAlloc((void **)&potential_pinned, zdim*ydim*xdim * sizeof(fptype), hipHostMallocMapped);
        checkCUDAError("hipHostMallocMapped");
        // Get the device pointers to the mapped memory
        hipHostGetDevicePointer((void **)&potential_gmem, (void *)potential_pinned, 0);
        checkCUDAError("hipHostGetDevicePointer");
    }
// reset potential before beginning
    #ifdef _OPENMP
    #pragma omp parallel for
    #endif
    for(int i = 0; i < zdim*ydim*xdim; i++)
        potential_pinned[i] = 0;

// reset potential before beginning
    #ifdef _OPENMP
    #pragma omp parallel for
    #endif
    for(int i = 0; i < zdim*ydim*xdim; i++)
        potential[i] = 0;

// for each charge layer in zdim
    for (int zc = 0; zc < zdim; zc++) {
        if(verbose_level >= 3)
            printf("  FMM: charge layer %d\n", zc);
        fprintf(paniclog, "# FMM:   charge layer %d\n", zc);
        fflush(NULL);
        // call the actual function
        status |= fmm_bfs(charge+zc*ydim*xdim, potential_pinned, potential_gmem, root, logN, logN, P, xdim, ydim, zdim, zc, paniclog, use_gpu, verbose_level);

        if(status) return EXIT_FAILURE;
        // root->grow();
    }

    memcpy(potential, potential_pinned, zdim*ydim*xdim*sizeof(fptype));

    if(status) return EXIT_FAILURE;
    status |= gettimeofday(&time2, NULL);
    deltatime = (time2.tv_sec + time2.tv_usec/1e6) - (time1.tv_sec + time1.tv_usec/1e6);
    if(verbose_level >= 0)
        printf("FMM: took %f seconds\n", deltatime);
    fflush(NULL);

// closing
    fclose(paniclog);
    // delete root;
    free(tree);
    first_time = 0;
    return status ? EXIT_FAILURE : EXIT_SUCCESS;
}


// Exact O(N^2) calculation of potential
void calc_potential_exact( const fptype *charge,
                        const int xdim, const int ydim, const int zdim,
                        fptype *potential, int use_gpu)
{
    int status = 0;
    timeval time1, time2;
    status |= gettimeofday(&time1, NULL);
    if(use_gpu)
        calc_potential_exact_gpu(charge, xdim, ydim, zdim, potential);
    else
    {
    // reset potential before beginning
        #ifdef _OPENMP
        #pragma omp parallel for
        #endif
        for(int i = 0; i < zdim*ydim*xdim; i++)
            potential[i] = 0;

        for(int z_ = 0; z_ < zdim; z_++) {  // source loop
            for(int y_ = 0; y_ < ydim; y_++) {
                for(int x_ = 0; x_ < xdim; x_++) {
                    fptype q = charge[z_*ydim*xdim + y_*xdim + x_];
                    if(q == 0) continue;
                    for(int z = 0; z < zdim; z++) { // observation point loop
                        #ifdef _OPENMP
                        #pragma omp parallel for
                        #endif
                        for(int y = 0; y < ydim; y++) {
                            for(int x = 0; x < xdim; x++) {
                                if(z == z_ && y == y_ && x == x_) continue;    // skip on itself
                                Vector3 R(x-x_, y-y_, z-z_);
                                potential[z*ydim*xdim + y*xdim + x] += q / R.magnitude();
                            }
                        }
                    }
                }
            }
        }
    }
    status |= gettimeofday(&time2, NULL);
    double deltatime = (time2.tv_sec + time2.tv_usec/1e6) - (time1.tv_sec + time1.tv_usec/1e6);
    if(0)
        printf("Exact: took %f seconds\n", deltatime);
    fflush(NULL);
}


// H field based on nearest neighbor coupling only
void calc_H_nearest_neighbor(   const Vector3 *M, Vector3 *H,
                                const int xdim, const int ydim, const int zdim )
{
    for(int i = 0; i < zdim*ydim*xdim; i++)
        H[i] = -0.2 * (   ((i-xdim >= 0)        ? M[i-xdim] : Vector3(0,0,0))     // top
                        + ((i+xdim < ydim*xdim) ? M[i+xdim] : Vector3(0,0,0))     // bottom
                        + ((i%xdim != 0)        ? M[i-1]    : Vector3(0,0,0))     // left
                        + (((i+1)%xdim != 0)    ? M[i+1]    : Vector3(0,0,0)) );  // right
}
